#include "hip/hip_runtime.h"
#include "Loop.h"
#include <stdio.h>
#include "mycuda.cuh"
#include <iostream>



extern "C" void loop_free(void *p)
{
	CUDACALL(hipFree(p));
}

extern "C" void GENDATA( void *p){
	unsigned a[2] = {1,1};

	CUDACALL( hipMemcpy(p , a , 2*sizeof(unsigned) , hipMemcpyHostToDevice));
}


extern "C" void* loop_malloc( unsigned n )
{
	unsigned *i;
	CUDACALL(hipMalloc( (void**)&i, sizeof(int) * n));
	return i;
}

typedef void (*loop_kernal_func)(void*,unsigned,unsigned) ;

__global__ void loop_helper( loop_kernal_func* f,
				void* arg, unsigned arg_bytes)
{
	 unsigned i = CUDAINDEX;

	 (*f)( arg , arg_bytes, i);

}



extern "C" void loop_exec( void (*loop_kernal)(void*,unsigned,unsigned),
				void* arg, unsigned arg_bytes,
				unsigned n)
{
	loop_kernal_func *h_f , *d_f;
	//this should be constant memory.
	h_f = (loop_kernal_func*)malloc(sizeof(loop_kernal_func));
	h_f[0] = loop_kernal;
	CUDACALL(hipMalloc((void**)&d_f,sizeof(loop_kernal_func)));

	std::cout<< "Here1"<<std::endl;
	CUDACALL(hipMemcpy( d_f , h_f , sizeof(loop_kernal_func) ,hipMemcpyHostToDevice ));
	std::cout<< "Here2"<<std::endl;
	CUDALAUNCH( loop_helper , n , (d_f,arg, arg_bytes));
	std::cout<< "Here3"<<std::endl;
	//CUDALAUNCH( (*loop_kernal) , n , (arg, arg_bytes,-1));
	//not sure what else to put. I really would like to work in a class for this part
	// Going to have to talk it out via email... I guess
}








