#include "Loop.h"
#include <stdio.h>
#include "mycuda.cuh"




extern "C" void loop_free(void *p)
{
	CUDACALL(hipFree(p));
}


extern "C" void* loop_malloc( unsigned n)
{
	unsigned *i;
	CUDACALL(hipMalloc(&i, sizeof(int) * n));
	return i;
}

extern "C" void loop_exec( void (*loop_kernal)(void*,unsigned,unsigned),
				void* arg, unsigned arg_bytes,
				unsigned n)
{
	(*loop_kernal)<<<ceildiv((n), BLOCK_SIZE),BLOCK_SIZE>>>(arg, arg_bytes , -1);
	//not sure what else to put. I really would like to work in a class for this part
	// Going to have to talk it out via email... I guess
}
