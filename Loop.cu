#include "Loop.h"
#include <stdio.h>
#include "mycuda.cuh"




extern "C" void loop_free(void *p)
{
	CUDACALL(hipFree(p));
}

extern "C" void GENDATA( void *p){
	unsigned a[2] = {1,1};

	CUDACALL( hipMemcpy(p , a , 2*sizeof(unsigned) , hipMemcpyHostToDevice));
}


extern "C" void* loop_malloc( unsigned n )
{
	unsigned *i;
	CUDACALL(hipMalloc( (void**)&i, sizeof(int) * n));
	return i;
}

extern "C" void loop_exec( void (*loop_kernal)(void*,unsigned,unsigned),
				void* arg, unsigned arg_bytes,
				unsigned n)
{
	//(*loop_kernal)<<<2,1>>>(arg, arg_bytes , -1);
	CUDALAUNCH( (*loop_kernal) , n , (arg, arg_bytes, -1));
	//not sure what else to put. I really would like to work in a class for this part
	// Going to have to talk it out via email... I guess
}








