#include "hip/hip_runtime.h"
#include "Loop.h"
#include <stdio.h>
#include "mycuda.cuh"
#include <iostream>



extern "C" void loop_free(void *p)
{
	CUDACALL(hipFree(p));
}

extern "C" void GENDATA( void *p){
	unsigned a[2] = {1,1};

	CUDACALL( hipMemcpy(p , a , 2*sizeof(unsigned) , hipMemcpyHostToDevice));
}


extern "C" void* loop_malloc( unsigned n )
{
	unsigned *i;
	CUDACALL(hipMalloc( (void**)&i, sizeof(int) * n));
	return i;
}

typedef void (*loop_kernal_func)(void*,unsigned,unsigned) ;

template<class O> __global__
void loop_helper( O op,
				void* arg, unsigned arg_bytes)
{
	 unsigned i = CUDAINDEX;

	 op( arg , arg_bytes, i);

}



template<class O>  void loop_exec( O op,
				void* arg, unsigned arg_bytes,
				unsigned n)
{
	loop_kernal_func *h_f , *d_f;
	//this should be constant memory.

	/*
	h_f = (loop_kernal_func*)malloc(sizeof(loop_kernal_func));
	h_f[0] = loop_kernal;
	CUDACALL(hipMalloc((void**)&d_f,sizeof(loop_kernal_func)));

	//std::cout<< "Here1"<<std::endl;
	CUDACALL(hipMemcpy( d_f , h_f , sizeof(loop_kernal_func) ,hipMemcpyHostToDevice ));
	//std::cout<< "Here2"<<std::endl;
	*/




	CUDACALL(hipMallocManaged(&h_f , sizeof(loop_kernal_func)));
	//h_f[0] = loop_kernal;


	//CUDALAUNCH( loop_helper , n , (h_f,arg, arg_bytes));

	loop_helper<<< ceildiv((n), 256), 256 >>>(O(),arg, arg_bytes);

	//std::cout<< "Here3"<<std::endl;

	//CUDALAUNCH( (*loop_kernal) , n , (arg, arg_bytes,-1));
	//not sure what else to put. I really would like to work in a class for this part
	// Going to have to talk it out via email... I guess
}








