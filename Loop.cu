#include "hip/hip_runtime.h"
#include "Loop.h"
#include <stdio.h>
#include "mycuda.cuh"




extern "C" void loop_free(void *p)
{
	CUDACALL(hipFree(p));
}

extern "C" void GENDATA( void *p){
	unsigned a[] = {1,1};

	CUDACALL( hipMemcpy(p , a , 2*sizeof(unsigned) , hipMemcpyHostToDevice));
}


extern "C" void* loop_malloc( unsigned n )
{
	unsigned *i;
	CUDACALL(hipMalloc( (void**)&i, sizeof(int) * n));
	return i;
}

extern "C" void loop_exec( void (*loop_kernal)(void*,unsigned,unsigned),
				void* arg, unsigned arg_bytes,
				unsigned n)
{
	(*loop_kernal)<<<ceildiv((n), BLOCK_SIZE),BLOCK_SIZE>>>(arg, arg_bytes , -1);
	//not sure what else to put. I really would like to work in a class for this part
	// Going to have to talk it out via email... I guess
}



__global__ void Run_Me( int* The_Array , int size)
{
	int ID = blockIdx.x;
	if(ID < 4)
	The_Array[ID] = The_Array[ID] * The_Array[ID];

}




