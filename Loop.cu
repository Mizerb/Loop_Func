#include "Loop.h"
#include <stdio.h>
#include "mycuda.cuh"




extern "C" void loop_free(void *p)
{
	CUDACALL(hipFree(p));
}


extern "C" void* loop_malloc( unsigned n)
{
	unsigned *i;
	CUDACALL(hipMalloc(&i, sizeof(int) * n));
	return i;
}

