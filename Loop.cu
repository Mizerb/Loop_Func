#include "Loop.h"
#include <stdio.h>
#include "mycuda.cuh"




extern "C" void loop_free(void *p)
{
	CUDACALL(hipFree(p));
}


extern "C" void* loop_malloc( unsigned n)
{
	unsigned *i;
	CUDACALL(hipMalloc(&i, sizeof(int) * n));
	return i;
}

extern "C" void loop_exec( void (*loop_kernal)(void*,unsigned),
				void* arg, unsigned arg_bytes,
				unsigned n)
{
	dim3 Block ( 4 ,  1);


	(*loop_kernal)<<<n,BLOCK_SIZE>>>(arg, arg_bytes);



}
