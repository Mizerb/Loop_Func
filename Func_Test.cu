#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "Loop.h"
#include "mycuda.cuh"


#include <stdio.h>

__device__ void Run_Me( void* INPUT, unsigned  n , unsigned size)
{
	struct arg * The_Array = (struct arg*) INPUT;

	int ID = size;
	if(ID < 2){
	The_Array->c[ID] = The_Array->a[ID] + The_Array->b[ID];
	}
}



void Test( thrust::device_vector<int> &A , void(*f)(int*,int) )
{
	int * GG = thrust::raw_pointer_cast(&A[0]);

	std::cout<<"Stalling"<<std::endl;

	dim3 Block ( 4 ,  1);
	(*f)<<<Block,1>>>(GG, 4);

	std::cout<<"this is the silliest thing, I have every done"<<std::endl;
}

template <typename T>
void * arg_pass( T &a )
{
	T *i;	// So Nasty
	CUDACALL(hipMalloc( (void**)&i, sizeof(T) ));
	CUDACALL( hipMemcpy(i , &a , sizeof(T) , hipMemcpyHostToDevice));
	return i;
}

template <typename T>
void * Something( T &a)
{
	hipMemcpyToSymbol(HIP_SYMBOL( &a ), sizeof(T));
}


int main()
{
	hipDeviceReset();

	struct arg a;
	a.a  = (unsigned int*)loop_malloc( 2);
	a.b  = (unsigned int*)loop_malloc( 2);
	a.c =  (unsigned int*)loop_malloc(2);

	std::cout << a.a << std::endl;

	unsigned int *d = new unsigned int[2];

	std::cout<< "ARG is" <<&a <<std::endl;

	GENDATA(a.a);
	GENDATA(a.b);



	loop_exec( Run_Me , arg_pass(a) , 2 , 2);

	hipDeviceSynchronize();

	std::cout << a.a << std::endl;

	CUDACALL( hipMemcpy(d , a.c , 2*sizeof(unsigned) , hipMemcpyDeviceToHost));

	hipDeviceReset();

	printf( "%d , %d\n", d[0] , d[1]);



	return 0;

}

