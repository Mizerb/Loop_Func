#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "Loop.h"
#include "mycuda.cuh"


#include <stdio.h>



#define LOOP_EXEC(func,arg, arg_bytes,n) loop_helper<<<ceildiv(n,256),256>>>(func, arg, arg_bytes)




class Runme {
public:
	__device__ void operator() ( void* INPUT, unsigned n ,unsigned size)
	{
		struct arg * The_Array = (struct arg*) INPUT;

		int ID = size;
		if(ID < 2){
			The_Array->c[ID] = The_Array->a[ID] + The_Array->b[ID];
		}
	}
};



void Test( thrust::device_vector<int> &A , void(*f)(int*,int) )
{
	int * GG = thrust::raw_pointer_cast(&A[0]);

	std::cout<<"Stalling"<<std::endl;

	dim3 Block ( 4 ,  1);
	(*f)<<<Block,1>>>(GG, 4);

	std::cout<<"this is the silliest thing, I have every done"<<std::endl;
}

template <typename T>
void * arg_pass( T &a )
{
	T *i;	// So Nasty
	CUDACALL(hipMalloc( (void**)&i, sizeof(T) ));
	CUDACALL( hipMemcpy(i , &a , sizeof(T) , hipMemcpyHostToDevice));
	return i;
}

template <typename T>
void * Something( T &a)
{
	hipMemcpyToSymbol(HIP_SYMBOL( &a ), sizeof(T));
}


int main()
{
	hipDeviceReset();

	struct arg a;
	a.a  = (unsigned int*)loop_malloc( 2);
	a.b  = (unsigned int*)loop_malloc( 2);
	a.c =  (unsigned int*)loop_malloc(2);

	std::cout << a.a << std::endl;

	unsigned int *d = new unsigned int[2];

	std::cout<< "ARG is" <<&a <<std::endl;

	GENDATA(a.a);
	GENDATA(a.b);



	std::cout << "HERE" <<std::endl;
	//loop_exec( Runme() , arg_pass(a) , 2 , 2);
	LOOP_EXEC(Runme(), arg_pass(a), 2 ,2);

	std::cout<< "WHAT THE" << std::endl;

	hipDeviceSynchronize();

	std::cout << a.a << std::endl;

	CUDACALL( hipMemcpy(d , a.c , 2*sizeof(unsigned) , hipMemcpyDeviceToHost));



	//printf( "%d , %d\n", d[0] , d[1]);

	std::cout<< d[0] << " , " << d[1] <<std::endl;

	return 0;

}

typedef void (*loop_kernal_func)(void*,unsigned,unsigned) ;



template<class O> __global__
void loop_helper( O op,
				void* arg, unsigned arg_bytes)
{
	 unsigned i = CUDAINDEX;

	 op( arg , arg_bytes, i);

}



template<class O>  void loop_exec( O op,
				void* arg, unsigned arg_bytes,
				unsigned n)
{
	loop_kernal_func *h_f , *d_f;
	//this should be constant memory.

	/*
	h_f = (loop_kernal_func*)malloc(sizeof(loop_kernal_func));
	h_f[0] = loop_kernal;
	CUDACALL(hipMalloc((void**)&d_f,sizeof(loop_kernal_func)));

	//std::cout<< "Here1"<<std::endl;
	CUDACALL(hipMemcpy( d_f , h_f , sizeof(loop_kernal_func) ,hipMemcpyHostToDevice ));
	//std::cout<< "Here2"<<std::endl;
	*/




	CUDACALL(hipMallocManaged(&h_f , sizeof(loop_kernal_func)));
	//h_f[0] = loop_kernal;


	//CUDALAUNCH( loop_helper , n , (h_f,arg, arg_bytes));

	loop_helper<<< ceildiv((n), 256), 256 >>>(O(),arg, arg_bytes);

	//std::cout<< "Here3"<<std::endl;

	//CUDALAUNCH( (*loop_kernal) , n , (arg, arg_bytes,-1));
	//not sure what else to put. I really would like to work in a class for this part
	// Going to have to talk it out via email... I guess
}

