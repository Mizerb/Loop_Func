#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "Loop.h"
#include "mycuda.cuh"


#include <stdio.h>

__global__ void Run_Me( void* INPUT, unsigned  n , unsigned size)
{
	struct arg * The_Array = (struct arg*) INPUT;

	int ID =CUDAINDEX;
	if(ID < 2){
	The_Array->c[ID] = The_Array->a[ID] + The_Array->b[ID];
	}
}



void Test( thrust::device_vector<int> &A , void(*f)(int*,int) )
{
	int * GG = thrust::raw_pointer_cast(&A[0]);

	std::cout<<"Stalling"<<std::endl;

	dim3 Block ( 4 ,  1);
	(*f)<<<Block,1>>>(GG, 4);

	std::cout<<"this is the silliest thing, I have every done"<<std::endl;
}



int main()
{
	hipDeviceReset();

	struct arg a;
	a.a  = (unsigned int*)loop_malloc( 2);
	a.b  = (unsigned int*)loop_malloc( 2);
	a.c =  (unsigned int*)loop_malloc(2);

	std::cout << a.a << std::endl;

	unsigned int *d = new unsigned int[2];

	std::cout<< "ARG is" <<&a <<std::endl;

	GENDATA(a.a);
	GENDATA(a.b);


	loop_exec( Run_Me , &a , 2 , 2);

	hipDeviceSynchronize();

	std::cout << a.a << std::endl;

	CUDACALL( hipMemcpy(d , a.c , 2*sizeof(unsigned) , hipMemcpyDeviceToHost));

	hipDeviceReset();

	printf( "%d , %d\n", d[0] , d[1]);



	return 0;

}

