#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "Loop.h"



#include <stdio.h>

__global__ void Run_Me( int* The_Array , int size)
{
	int ID = blockIdx.x;
	if(ID < 4)
	The_Array[ID] = The_Array[ID] * The_Array[ID];

}



void Test( thrust::device_vector<int> &A , void(*f)(int*,int) )
{
	int * GG = thrust::raw_pointer_cast(&A[0]);

	std::cout<<"Stalling"<<std::endl;

	dim3 Block ( 4 ,  1);
	(*f)<<<Block,1>>>(GG, 4);

	std::cout<<"this is the silliest thing, I have every done"<<std::endl;
}


/*
int main()
{
	thrust::host_vector<int> C(4);

	C[0] = 1;
	C[1] = 2;
	C[2] = 3;
	C[3] = 4;

	/*
	std::cout<<"RUnning"<<std::endl;
	thrust::device_vector<int> A = C;

	Test(A, Run_Me);

	for(int i = 0 ; i< 4 ;i++)
	{
		std::cout << A[i] <<std::endl;
	}
	char wait;

	int *x; int a = 0;
	x = &a;

	loop_free( x);
	int wait;
	std::cin >> wait;

}
*/
